/*
  This program is imported from:
  https://gist.github.com/andrejbauer/7919569

  This program is an adaptation of the Mandelbrot program
  from the Programming Rosetta Stone, see
  http://rosettacode.org/wiki/Mandelbrot_set

  See http://www.imagemagick.org/Usage/color_mods/ for what ImageMagick
  can do. It can do a lot.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#define BX 32
#define BY 32

// CUDA kernel. Each thread takes care of one element of c
__global__ void mandelbrot_kernel(double *result, double dx, double dy, double xmin, double ymax, int maxiter, int n){

    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if ((tx < n) && (ty < n)){
        // printf(" blockz: %d, blocky: %d, blockx: %d, threadidz: %d, threadidy: %d, threadidx: %d.\n", blockIdx.z, blockIdx.y, blockIdx.x, threadIdx.z, threadIdx.y, threadIdx.x);    
        double y = ymax - ty * dy;
        double x = xmin + tx * dx;
        // printf("threadidy: %d, threadidx: %d, y: %f, x: %f.\n", threadIdx.y, threadIdx.x, y, x);

        double u = 0.0;
        double v = 0.0;
        double u2 = u * u;
        double v2 = v * v;

        int k;
        for (k = 1; k < maxiter && (u2 + v2 < 4.0); k++){
            v = 2 * u * v + y;
            u = u2 - v2 + x;
            u2 = u * u;
            v2 = v * v;
        };
        /* compute result */
        if (k >= maxiter) {
            result[ty * n + tx] = 0;
        }
        else {
            result[ty * n + tx] = 1;
            // printf("tx: %i, ty: %i at k: %i", tx, ty, k);
        };
        __syncthreads();
    }
}

int main(int argc, char* argv[])
{
    const char* filename;
    uint16_t maxiter;
    int xres;
    /* Parse the command line arguments. */
    if (argc == 2){
        /* The output file name */
        filename = argv[1];
        /* Maximum number of iterations, at most 65535. */
        maxiter = 1000;
        xres = 1000;
    }
    else if (argc == 4){
        /* The output file name */
        filename = argv[1];
        /* Maximum number of iterations, at most 65535. */
        maxiter = atoi(argv[2]);
        xres = atoi(argv[3]);
    }
    else{
        printf("Usage:   %s <out.ppm>\n", argv[0]);
        printf("Example: %s pic.ppm\n", argv[0]);
        return -1;
    }

    /* The window in the plane. */
    const double xmin = -2;
    const double xmax = 1;
    const double ymin = -1.5;
    const double ymax = 1.5;

    /* Image size, width is given, height is computed. */
    const int yres = xres;

    //Host output vector
    double *h_result;
    //Device output vector
    double *d_result;

    // Size, in bytes, of each vector
    size_t bytes = xres*yres*sizeof(double);

    // allocate memory
    h_result = (double*)malloc(bytes);
    hipMalloc(&d_result, bytes);

    /* Precompute pixel width and height. */
    double dx=(xmax-xmin)/xres;
    double dy=(ymax-ymin)/yres;
 
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    dim3 block(BX,BY);
    dim3 grid((xres+block.x-1)/block.x, (yres+block.y-1)/block.y);
    // printf("blockx: %i, blocky: %i, gridx: %i, gridy: %i\n", block.x, block.y, grid.x, grid.y);


    // Execute the kernel
    mandelbrot_kernel <<<grid, block>>>(d_result, dx, dy, xmin, ymax, maxiter, xres);

    hipMemcpy( h_result, d_result, bytes, hipMemcpyDeviceToHost );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Elapsed time:  %3.3f ms \n", time);

    /* Open the file and write the header. */
    FILE * fp = fopen(filename, "wb");

    /*write ASCII header to the file*/
    fprintf(fp,
            "P6\n# Mandelbrot, xmin=%lf, xmax=%lf, ymin=%lf, ymax=%lf, maxiter=%d\n%d\n%d\n%d\n",
            xmin, xmax, ymin, ymax, maxiter, xres, yres, (maxiter < 256 ? 256 : maxiter));

    unsigned char color[] = {1,3,3,5,3,3};
    const unsigned char black[] = {0, 0, 0, 0, 0, 0};
    for (int c = 0; c < yres*xres; c++){
        if (h_result[c] == 0) fwrite(black, 6, 1, fp);
        else fwrite(color, 6, 1, fp);
    }
    fclose(fp);
    return 0;
}
